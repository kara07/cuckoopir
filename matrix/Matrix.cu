#include "hip/hip_runtime.h"
#include "Matrix.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>

#define GET_ELEMENT(i, j, m, n) (i * n + j)
#define CHECK(call) \
{ \
    hipError_t cudaStatus = (call); \
    if(cudaStatus != hipSuccess) \
    { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(cudaStatus)); \
        exit(1); \
    } \
}

extern "C"
{

    __global__ void mat_mul(int *mat1, int *mat2, int *result) {
        const int bid = blockIdx.x;
        const int tid = threadIdx.x;
        // 每个线程计算一行
        const int row = bid * THREAD_NUM + tid;
        for (int c = 0; c < R_SIZE; c++) {
            for (int n = 0; n < R_SIZE; n++) {
            result[row*R_SIZE+c] += mat1[row*R_SIZE+n] * mat2[n*R_SIZE+c];
            }
        }
    }


    void generate_random_matrix(struct Matrix_int *mat, int m, int n)
    {
        assert(mat != NULL);
        mat->width = n;
        mat->height = m;
        if (mat->elements == NULL)
        {
            mat->elements = (int *)malloc(m * n * sizeof(int));
        }

        for(int i = 0; i < n; ++i)
        {
            for(int j = 0; j < m; ++j)
            {
                mat->elements[i * n + j] = rand() % 100;
            }
        }
    }

    void destroy_matrix(struct Matrix_int *mat)
    {
        assert(mat != NULL);
        free(mat->elements);
    }

    void MatrixMulCPU(int *a, int *b, int *c, int m, int n, int p)
    {
        for(int i = 0; i < m; ++i)
        {
            for(int j = 0; j < p; ++j)
            {
                for(int k = 1; k <= n; ++k)
                {
                    c[i*n + j] +=  a[ i*n +k - 1] * b[(k - 1)*p  + j ];
                }
            }
        }
    }


}

__global__ void mm_kernel(int *mat_1, int *mat_2, int *mat_3, int m, int n, int p)
{
    __shared__ int mat_1_tile[BLOCK_DIM][BLOCK_DIM];
    __shared__ int mat_2_tile[BLOCK_DIM][BLOCK_DIM];

    int acc_sum{0};

    for (size_t tile_idx{0};
         tile_idx < ceilf(static_cast<float>(n) / BLOCK_DIM); ++tile_idx)
    {
        size_t i{blockIdx.y * blockDim.y + threadIdx.y};
        size_t j{tile_idx * blockDim.x + threadIdx.x};
        if ((i < m) && (j < n))
        {
            mat_1_tile[threadIdx.y][threadIdx.x] = mat_1[i * n + j];
        }
        else
        {
            mat_1_tile[threadIdx.y][threadIdx.x] = 0;
        }
        i = tile_idx * blockDim.y + threadIdx.y;
        j = blockIdx.x * blockDim.x + threadIdx.x;
        if ((i < n) && (j < p))
        {
            mat_2_tile[threadIdx.y][threadIdx.x] = mat_2[i * p + j];
        }
        else
        {
            mat_2_tile[threadIdx.y][threadIdx.x] = 0;
        }
        __syncthreads();
        for (size_t k{0}; k < BLOCK_DIM; ++k)
        {
            acc_sum += mat_1_tile[threadIdx.y][k] * mat_2_tile[k][threadIdx.x];
        }
        __syncthreads();
    }

    // 2D block and 2D thread
    // Each thread computes one cell in mat_3.
    size_t i{blockIdx.y * blockDim.y + threadIdx.y};
    size_t j{blockIdx.x * blockDim.x + threadIdx.x};

    if ((i < m) && (j < p))
    {
        mat_3[i * p + j] = acc_sum;
    }
}


    void MatrixMul(Matrix_int *a, Matrix_int *b, Matrix_int *c, int m, int n, int k)
    {
        assert(a != NULL && b != NULL && c != NULL);
        assert(a->elements != NULL && b->elements != NULL && c->elements != NULL);
        dim3 blockNum(32, 32);
        dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);
        int *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;
        int size_a = (a->width) * (a->height) * sizeof(int);
        int size_b = (b->width) * (b->height) * sizeof(int);
        int size_c = (c->width) * (c->height) * sizeof(int);

        CHECK(hipMalloc((void**)&d_a, size_a));
        CHECK(hipMalloc((void**)&d_b, size_b));
        CHECK(hipMalloc((void**)&d_c, size_c));

        hipMemcpy(d_a, a->elements, size_a, ::hipMemcpyHostToDevice);
        hipMemcpy(d_b, b->elements, size_b, ::hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        mm_kernel<<<blockNum, threadsPerBlock>>>((int *)d_a, (int *)d_b, (int *)d_c, m, n, k);
        hipMemcpy(c->elements, d_c, size_c, ::hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }


    void test()
    {
        Matrix_int a {1000, 1000, NULL};
        Matrix_int b {1000, 1000, NULL};
        Matrix_int c {1000, 1000, NULL};

        generate_random_matrix(&a, a.height, a.width);
        generate_random_matrix(&b, b.height, b.width);
        generate_random_matrix(&c, c.height, c.width);

        // auto start_time_cpu = std::chrono::high_resolution_clock::now();
        // MatrixMulCPU(a.elements, b.elements, c.elements, 1024, 1024, 1024);
        // auto end_time_cpu = std::chrono::high_resolution_clock::now();
        // auto time_elasped = end_time_cpu - start_time_cpu;
        // std::cout << "CPU duration: " << std::chrono::duration_cast<std::chrono::microseconds>(time_elasped).count() << std::endl;

        auto start_time_gpu = std::chrono::high_resolution_clock::now();
        MatrixMul(&a, &b, &c, 8192, 8192, 8192);
        auto end_time_gpu = std::chrono::high_resolution_clock::now();
        auto time_elasped_gpu = end_time_gpu - start_time_gpu;
        std::cout << "GPU duration: " << std::chrono::duration_cast<std::chrono::microseconds>(time_elasped_gpu).count() << std::endl;

        int *mat1, *mat2, *result;
        int *g_mat1, *g_mat2, *g_mat_result;

        mat1 = (int*) malloc(M_SIZE * sizeof(int));
        mat2 = (int*) malloc(M_SIZE * sizeof(int));
        result = (int*) malloc(M_SIZE * sizeof(int));

        // initialize
        for (int i = 0; i < M_SIZE; i++) {
        mat1[i] = rand()/1000000;
        mat2[i] = rand()/1000000;
        result[i] = 0;
        }

        hipMalloc((void **)&g_mat1, sizeof(int) * M_SIZE);
        hipMalloc((void **)&g_mat2, sizeof(int) * M_SIZE);
        hipMalloc((void **)&g_mat_result, sizeof(int) * M_SIZE);

        hipMemcpy(g_mat1, mat1, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
        hipMemcpy(g_mat2, mat2, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);

        auto start_time_gpu2 = std::chrono::high_resolution_clock::now();
        mat_mul<<<BLOCK_NUM, THREAD_NUM>>>(g_mat1, g_mat2, g_mat_result);
        auto end_time_gpu2 = std::chrono::high_resolution_clock::now();
        auto time_elasped_gpu2 = end_time_gpu2 - start_time_gpu2;
        std::cout << "GPU duration_2: " << std::chrono::duration_cast<std::chrono::microseconds>(time_elasped_gpu2).count() << std::endl;



        hipMemcpy(result, g_mat_result, sizeof(int) * M_SIZE, hipMemcpyDeviceToHost);

    }

    void test2(){
        int *mat1, *mat2, *result;
        int *g_mat1, *g_mat2, *g_mat_result;
    
        mat1 = (int*) malloc(M_SIZE * sizeof(int));
        mat2 = (int*) malloc(M_SIZE * sizeof(int));
        result = (int*) malloc(M_SIZE * sizeof(int));

        // initialize
        for (int i = 0; i < M_SIZE; i++) {
        mat1[i] = rand()/1000000;
        mat2[i] = rand()/1000000;
        result[i] = 0;
        }

        hipMalloc((void **)&g_mat1, sizeof(int) * M_SIZE);
        hipMalloc((void **)&g_mat2, sizeof(int) * M_SIZE);
        hipMalloc((void **)&g_mat_result, sizeof(int) * M_SIZE);

        hipMemcpy(g_mat1, mat1, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
        hipMemcpy(g_mat2, mat2, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);

        mat_mul<<<BLOCK_NUM, THREAD_NUM>>>(g_mat1, g_mat2, g_mat_result);

        hipMemcpy(result, g_mat_result, sizeof(int) * M_SIZE, hipMemcpyDeviceToHost);
    }

int main()
{
    test();
}
